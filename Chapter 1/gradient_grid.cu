#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>
#include "../Common/ImageWriter.cuh"

const int ImageWidth = 1024;
const int ImageHeight = 512;
const int BlockSize = 1;

struct PixelColor
{
    float r;
    float g;
    float b;
};

// Kernel function to add the elements of two arrays
__global__
void CalculatePixelColors(int width, int height, PixelColor* pixels)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int numPixels = width * height;
    for (int k = index; k < numPixels; k += stride)
    {
        int j =  height - 1 - (k / width);
        int i = k % width;
        pixels[k].r = static_cast<float>(i) / static_cast<float>(width);
        pixels[k].g = static_cast<float>(j) / static_cast<float>(height);
        pixels[k].b = 0.2;
    }
}

void SaveImage(std::string fileName, int width, int height, PixelColor* pixels)
{
  std::ofstream imageFile;
  imageFile.open(fileName.c_str());
  imageFile << "P3" << std::endl  << width << " " << height << std::endl << 255 << std::endl;
  int k = 0;
  for (int j = 0; j < height; ++j)
  {
      for (int i = 0; i < width; ++i)
      {
          int ir = static_cast<int>(255.99 * pixels[k].r);
          int ig = static_cast<int>(255.99 * pixels[k].g);
          int ib = static_cast<int>(255.99 * pixels[k].b);
          imageFile << ir << " " << ig << " " << ib << std::endl;
          k++;
      }
  }
  imageFile.close();
}

int main(int argc, char** argv)
{
  std::string fileName = ImageWriter::GetFileName(argc, argv);
  
  // Allocate Unified Memory – accessible from CPU or GPU
  int numPixels = ImageWidth*ImageHeight;
  PixelColor *pixels;
  hipMallocManaged(&pixels, numPixels*sizeof(PixelColor));

  // Run kernel on the GPU
  int numBlocks = (numPixels + BlockSize - 1) / BlockSize;
  CalculatePixelColors<<<numBlocks, BlockSize>>>(ImageWidth, ImageHeight, pixels);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  SaveImage(fileName, ImageWidth, ImageHeight, pixels);

  // Free memory
  hipFree(pixels);
  
  return 0;
}