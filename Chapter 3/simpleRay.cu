#include "hip/hip_runtime.h"
#include "../Common/ImageWriter.cuh"
#include "../Common/Ray.cuh"
#include "../Common/Vector3.cuh"

const int ImageWidth = 1024;
const int ImageHeight = 512;
const int BlockSize = 1;

__host__ __device__ 
Vector3 CalculateColor(const Ray &r)
{
    Vector3 unitDirection = UnitVector(r.Direction());
    float t = 0.5 * (unitDirection.Y() + 1.0);
    t = min(1.0, max(0.0, t));
    return (1.0 - t) * Vector3::One() + t * Vector3(0.5, 0.7, 1.0);
}

__global__
void CalculateImage(int width, int height, Vector3* pixels)
{
    const Vector3 lowerLeft(-2.0, -1.0, -1.0);
    const Vector3 horizontal(4.0, 0.0, 0.0);
    const Vector3 vertical(0.0, 2.0, 0.0);
    const Vector3 origin = Vector3::Zero();

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int numPixels = width * height;
    for (int k = index; k < numPixels; k += stride)
    {
        int j =  height - 1 - (k / width);
        int i = k % width;
        float u = static_cast<float>(i) / static_cast<float>(width);
        float v = static_cast<float>(j) / static_cast<float>(height);
        Ray ray(origin, lowerLeft + u * horizontal + v * vertical);
        pixels[k] = CalculateColor(ray);
    }
}

int main(int argc, char** argv)
{
  std::string fileName = ImageWriter::GetFileName(argc, argv);
  
  // Allocate Unified Memory – accessible from CPU or GPU
  int numPixels = ImageWidth*ImageHeight;
  Vector3 *pixels;
  hipMallocManaged(&pixels, numPixels*sizeof(Vector3));

  // Run kernel on the GPU
  int numBlocks = (numPixels + BlockSize - 1) / BlockSize;
  CalculateImage<<<numBlocks, BlockSize>>>(ImageWidth, ImageHeight, pixels);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  ImageWriter::WritePPM(fileName, ImageWidth, ImageHeight, pixels);

  // Free memory
  hipFree(pixels);
  
  return 0;
}